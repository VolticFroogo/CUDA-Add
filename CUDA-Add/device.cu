#include "hip/hip_runtime.h"
#include "device.cuh"

int* Device::Add(int* a, int* b, int n, size_t size)
{
	// Declare the device pointers.
	int* devA, * devB, * devC;

	// Allocate memory on the device.
	hipMalloc(&devA, size);
	hipMalloc(&devB, size);
	hipMalloc(&devC, size);

	// Copy A and B from the host onto the device.
	hipMemcpy(devA, a, size, hipMemcpyHostToDevice);
	hipMemcpy(devB, b, size, hipMemcpyHostToDevice);

	// Start the clock.
	auto start = std::chrono::high_resolution_clock::now();

	// Run the addition on the GPU (device).
	Kernel<<<31250, 1024>>>(devA, devB, devC, n);

	// Wait until all threads have finished.
	hipDeviceSynchronize();

	// Calculate the time of execution.
	auto finish = std::chrono::high_resolution_clock::now();
	auto delta = std::chrono::duration_cast<std::chrono::nanoseconds>(finish - start).count();

	// Print the time the device took to complete all of the addition.
	printf("Device exec time: %llu ns\n", delta);

	// Copy the values back from the device to the host.
	int* c = (int*)malloc(size);
	hipMemcpy(c, devC, size, hipMemcpyDeviceToHost);

	// Free all of the device's memory.
	hipFree(devA);
	hipFree(devB);
	hipFree(devC);

	return c;
}

__global__ void Device::Kernel(int* a, int* b, int* c, int n)
{
	int i = (blockIdx.x * blockDim.x) + threadIdx.x;

	// Set C to the sum of A and B.
	c[i] = a[i] + b[i];
}
