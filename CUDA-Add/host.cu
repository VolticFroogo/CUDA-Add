#include "hip/hip_runtime.h"
#include "host.cuh"

int* Host::Add(int* a, int* b, int n, size_t size)
{
	// Allocate memory for the sums on the host.
	int* c = (int*)malloc(size);

	// Start the clock.
	auto start = std::chrono::high_resolution_clock::now();

	// Iterate through every value in the array.
	for (int i = 0; i < n; i++)
	{
		// Set C to the sum of A and B.
		c[i] = a[i] + b[i];
	}

	// Calculate the time of execution.
	auto finish = std::chrono::high_resolution_clock::now();
	auto delta = std::chrono::duration_cast<std::chrono::nanoseconds>(finish - start).count();

	// Print the time the host took to complete all of the addition.
	printf("Host exec time:   %llu ns\n", delta);

	return c;
}
